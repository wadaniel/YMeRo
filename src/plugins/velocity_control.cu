#include "hip/hip_runtime.h"
#include "velocity_control.h"
#include "utils/simple_serializer.h"

#include <core/datatypes.h>
#include <core/pvs/particle_vector.h>
#include <core/pvs/views/pv.h>
#include <core/simulation.h>
#include <core/utils/cuda_common.h>
#include <core/utils/kernel_launch.h>

namespace VelocityControlKernels
{

inline __device__ bool is_inside(float3 r, float3 low, float3 high)
{
    return
        low.x <= r.x && r.x <= high.x &&
        low.y <= r.y && r.y <= high.y &&
        low.z <= r.z && r.z <= high.z;
}

__global__ void addForce(PVview view, DomainInfo domain, float3 low, float3 high, float3 force)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid >= view.size) return;

    Particle p;
    p.readCoordinate(view.particles, gid);
    float3 gr = domain.local2global(p.r);

    if (is_inside(gr, low, high))
        view.forces[gid] += make_float4(force, 0.0f);
}

__global__ void sumVelocity(PVview view, DomainInfo domain, float3 low, float3 high, float3 *totVel, int *nSamples)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    Particle p;
    
    p.u = make_float3(0.0f);

    if (gid < view.size) {

        p.read(view.particles, gid);
        float3 gr = domain.local2global(p.r);

        if (is_inside(gr, low, high))
            atomicAggInc(nSamples);
        else
            p.u = make_float3(0.0f);
    }

    float3 u = warpReduce(p.u, [](float a, float b) { return a+b; });
    
    if (__laneid() == 0 && dot(u, u) > 1e-8)
        atomicAdd(totVel, u);
}

} // namespace VelocityControlKernels

SimulationVelocityControl::SimulationVelocityControl(const YmrState *state, std::string name, std::vector<std::string> pvNames,
                                                     float3 low, float3 high,
                                                     int sampleEvery, int tuneEvery, int dumpEvery,
                                                     float3 targetVel, float Kp, float Ki, float Kd) :
    SimulationPlugin(state, name),
    pvNames(pvNames),
    low(low),
    high(high),
    currentVel(make_float3(0,0,0)),
    targetVel(targetVel),
    sampleEvery(sampleEvery),
    tuneEvery(tuneEvery),
    dumpEvery(dumpEvery), 
    force(make_float3(0, 0, 0)),
    pid(make_float3(0, 0, 0), Kp, Ki, Kd),
    accumulatedTotVel({0,0,0})
{}


void SimulationVelocityControl::setup(Simulation* simulation, const MPI_Comm& comm, const MPI_Comm& interComm)
{
    SimulationPlugin::setup(simulation, comm, interComm);

    for (auto &pvName : pvNames)
        pvs.push_back(simulation->getPVbyNameOrDie(pvName));
}

void SimulationVelocityControl::beforeForces(hipStream_t stream)
{
    for (auto &pv : pvs)
    {
        PVview view(pv, pv->local());
        const int nthreads = 128;

        SAFE_KERNEL_LAUNCH
            (VelocityControlKernels::addForce,
             getNblocks(view.size, nthreads), nthreads, 0, stream,
             view, state->domain, low, high, force );
    }
}

void SimulationVelocityControl::sampleOnePv(ParticleVector *pv, hipStream_t stream) {
    PVview pvView(pv, pv->local());
    const int nthreads = 128;
 
    SAFE_KERNEL_LAUNCH
        (VelocityControlKernels::sumVelocity,
         getNblocks(pvView.size, nthreads), nthreads, 0, stream,
         pvView, state->domain, low, high, totVel.devPtr(), nSamples.devPtr());
}

void SimulationVelocityControl::afterIntegration(hipStream_t stream)
{
    if (state->currentStep % sampleEvery == 0 && state->currentStep != 0)
    {
        debug2("Velocity control %s is sampling now", name.c_str());

        totVel.clearDevice(stream);
        for (auto &pv : pvs) sampleOnePv(pv, stream);
        totVel.downloadFromDevice(stream);
        accumulatedTotVel.x += totVel[0].x;
        accumulatedTotVel.y += totVel[0].y;
        accumulatedTotVel.z += totVel[0].z;
    }
    
    if (state->currentStep % tuneEvery != 0 || state->currentStep == 0) return;
    
    nSamples.downloadFromDevice(stream);
    nSamples.clearDevice(stream);
    
    long nSamples_loc, nSamples_tot = 0;
    double3 totVel_tot = make_double3(0,0,0);  

    nSamples_loc = nSamples[0];
    
    MPI_Check( MPI_Allreduce(&nSamples_loc,        &nSamples_tot, 1, MPI_LONG,   MPI_SUM, comm) );
    MPI_Check( MPI_Allreduce(&accumulatedTotVel,   &totVel_tot,   3, MPI_DOUBLE, MPI_SUM, comm) );

    currentVel = nSamples_tot ? make_float3(totVel_tot / nSamples_tot) : make_float3(0.f, 0.f, 0.f);
    force = pid.update(targetVel - currentVel);
    accumulatedTotVel = {0,0,0};
}

void SimulationVelocityControl::serializeAndSend(hipStream_t stream)
{
    if (state->currentStep % dumpEvery != 0 || state->currentStep == 0) return;

    waitPrevSend();
    SimpleSerializer::serialize(sendBuffer, state->currentTime, state->currentStep, currentVel, force);
    send(sendBuffer);
}




PostprocessVelocityControl::PostprocessVelocityControl(std::string name, std::string filename) :
    PostprocessPlugin(name)
{
    fdump = fopen(filename.c_str(), "w");
    if (!fdump) die("Could not open file '%s'", filename.c_str());
    fprintf(fdump, "# time time_step velocity force\n");
}

PostprocessVelocityControl::~PostprocessVelocityControl()
{
    fclose(fdump);
}

void PostprocessVelocityControl::deserialize(MPI_Status& stat)
{
    int currentTimeStep;
    TimeType currentTime;
    float3 vel, force;

    SimpleSerializer::deserialize(data, currentTime, currentTimeStep, vel, force);

    if (rank == 0) {
        fprintf(fdump,
                "%g %d "
                "%g %g %g "
                "%g %g %g\n",
                currentTime, currentTimeStep,
                vel.x, vel.y, vel.z,
                force.x, force.y, force.z
                );
        
        fflush(fdump);
    }
}
