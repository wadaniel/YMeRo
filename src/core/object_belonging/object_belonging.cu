#include "hip/hip_runtime.h"
#include "object_belonging.h"

#include <core/utils/kernel_launch.h>
#include <core/pvs/particle_vector.h>
#include <core/pvs/views/pv.h>
#include <core/pvs/object_vector.h>

#include <core/celllist.h>

__global__ void copyInOut(
        PVview view,
        const BelongingTags* tags,
        Particle* ins, Particle* outs,
        int* nIn, int* nOut)
{
    const int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid >= view.size) return;

    auto tag = tags[gid];
    const Particle p(view.particles, gid);

    if (tag == BelongingTags::Outside)
    {
        int dstId = atomicAggInc(nOut);
        if (outs) outs[dstId] = p;
    }

    if (tag == BelongingTags::Inside)
    {
        int dstId = atomicAggInc(nIn);
        if (ins)  ins [dstId] = p;
    }
}

ObjectBelongingChecker_Common::ObjectBelongingChecker_Common(const YmrState *state, std::string name) :
    ObjectBelongingChecker(state, name)
{}

ObjectBelongingChecker_Common::~ObjectBelongingChecker_Common() = default;


void ObjectBelongingChecker_Common::splitByBelonging(ParticleVector* src, ParticleVector* pvIn, ParticleVector* pvOut, hipStream_t stream)
{
    if (dynamic_cast<ObjectVector*>(src) != nullptr)
        error("Trying to split object vector %s into two per-particle, probably that's not what you wanted",
              src->name.c_str());

    if (pvIn != nullptr && typeid(*src) != typeid(*pvIn))
        error("PV type of inner result of split (%s) is different from source (%s)",
              pvIn->name.c_str(), src->name.c_str());

    if (pvOut != nullptr && typeid(*src) != typeid(*pvOut))
        error("PV type of outer result of split (%s) is different from source (%s)",
              pvOut->name.c_str(), src->name.c_str());

    {
        PrimaryCellList cl(src, 1.0f, state->domain.localSize);
        cl.build(stream);
        checkInner(src, &cl, stream);
    }

    info("Splitting PV %s with respect to OV %s. Number of particles: in/out/total %d / %d / %d",
         src->name.c_str(), ov->name.c_str(), nInside[0], nOutside[0], src->local()->size());

    // Need buffers because the source is the same as inside or outside
    PinnedBuffer<Particle> bufIn(nInside[0]), bufOut(nOutside[0]);

    nInside. clearDevice(stream);
    nOutside.clearDevice(stream);
    tags.downloadFromDevice(stream);

    PVview view(src, src->local());
    const int nthreads = 128;
    SAFE_KERNEL_LAUNCH(
            copyInOut,
            getNblocks(view.size, nthreads), nthreads, 0, stream,
            view,
            tags.devPtr(), bufIn.devPtr(), bufOut.devPtr(),
            nInside.devPtr(), nOutside.devPtr() );

    CUDA_Check( hipStreamSynchronize(stream) );

    if (pvIn  != nullptr)
    {
        int oldSize = (src == pvIn) ? 0 : pvIn->local()->size();
        pvIn->local()->resize(oldSize + nInside[0], stream);

        if (nInside[0] > 0)
            CUDA_Check( hipMemcpyAsync(pvIn->local()->coosvels.devPtr() + oldSize,
                    bufIn.devPtr(),
                    nInside[0] * sizeof(Particle),
                    hipMemcpyDeviceToDevice, stream) );


        info("New size of inner PV %s is %d", pvIn->name.c_str(), pvIn->local()->size());
        pvIn->cellListStamp++;
    }

    if (pvOut != nullptr)
    {
        int oldSize = (src == pvOut) ? 0 : pvOut->local()->size();
        pvOut->local()->resize(oldSize + nOutside[0], stream);

        if (nOutside[0] > 0)
            CUDA_Check( hipMemcpyAsync(pvOut->local()->coosvels.devPtr() + oldSize,
                    bufOut.devPtr(),
                    nOutside[0] * sizeof(Particle),
                    hipMemcpyDeviceToDevice, stream) );


        info("New size of outer PV %s is %d", pvOut->name.c_str(), pvOut->local()->size());
        pvOut->cellListStamp++;
    }
}

void ObjectBelongingChecker_Common::checkInner(ParticleVector* pv, CellList* cl, hipStream_t stream)
{
    tagInner(pv, cl, stream);

    nInside.clear(stream);
    nOutside.clear(stream);

    // Only count
    PVview view(pv, pv->local());
    const int nthreads = 128;
    SAFE_KERNEL_LAUNCH(
                copyInOut,
                getNblocks(view.size, nthreads), nthreads, 0, stream,
                view, tags.devPtr(), nullptr, nullptr,
                nInside.devPtr(), nOutside.devPtr() );

    nInside. downloadFromDevice(stream, ContainersSynch::Asynch);
    nOutside.downloadFromDevice(stream, ContainersSynch::Synch);

    say("PV %s belonging check against OV %s: in/out/total  %d / %d / %d",
        pv->name.c_str(), ov->name.c_str(), nInside[0], nOutside[0], pv->local()->size());
}

void ObjectBelongingChecker_Common::setup(ObjectVector* ov)
{
    this->ov = ov;
}

std::vector<std::string> ObjectBelongingChecker_Common::getChannelsToBeExchanged() const
{
    return {ChannelNames::motions};
}

ObjectVector* ObjectBelongingChecker_Common::getObjectVector()
{
    return ov;
}
