#include "hip/hip_runtime.h"
#include "membrane_extra_force.h"

#include <core/utils/kernel_launch.h>
#include <core/pvs/membrane_vector.h>
#include <core/pvs/views/ov.h>
#include <core/simulation.h>

#include <core/utils/cuda_common.h>

namespace MembraneExtraForcesKernels
{
__global__ void addForce(OVview view, const Force *forces)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid >= view.size) return;

    int locId = gid % view.objSize;

    view.forces[gid] += forces[locId].toFloat4();
}
} // namespace MembraneExtraForcesKernels

MembraneExtraForcePlugin::MembraneExtraForcePlugin(const YmrState *state, std::string name, std::string pvName, const PyTypes::VectorOfFloat3 &forces) :
    SimulationPlugin(state, name),
    pvName(pvName),
    forces(forces.size())
{
    HostBuffer<Force> hostForces(forces.size());

    for (int i = 0; i < forces.size(); ++i) {
        auto f = forces[i];
        hostForces.hostPtr()[i].f = make_float3(f[0], f[1], f[2]);
    }
    
    this->forces.copy(hostForces, 0);
}

void MembraneExtraForcePlugin::setup(Simulation *simulation, const MPI_Comm& comm, const MPI_Comm& interComm)
{
    SimulationPlugin::setup(simulation, comm, interComm);

    auto pv_ptr = simulation->getPVbyNameOrDie(pvName);
    if ( !(pv = dynamic_cast<MembraneVector*>(pv_ptr)) )
        die("MembraneExtraForcePlugin '%s' expects a MembraneVector (given '%s')", name.c_str(), pvName.c_str());
}

void MembraneExtraForcePlugin::beforeForces(hipStream_t stream)
{
    OVview view(pv, pv->local());
    const int nthreads = 128;

    SAFE_KERNEL_LAUNCH(
        MembraneExtraForcesKernels::addForce,
        getNblocks(view.size, nthreads), nthreads, 0, stream,
        view, forces.devPtr() );
}

