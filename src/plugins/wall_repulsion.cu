#include "hip/hip_runtime.h"
#include "wall_repulsion.h"

#include <core/pvs/particle_vector.h>
#include <core/pvs/views/pv.h>
#include <core/simulation.h>
#include <core/utils/cuda_common.h>
#include <core/utils/cuda_rng.h>
#include <core/utils/kernel_launch.h>
#include <core/walls/simple_stationary_wall.h>

namespace ChannelNames
{
static const std::string      sdf =      "sdf";
static const std::string grad_sdf = "grad_sdf";
} // namespace ChannelNames

namespace WallRepulsionPluginKernels
{
__global__ void forceFromSDF(PVview view, float* sdfs, float3* gradients, float C, float h, float maxForce)
{
    int pid = blockIdx.x * blockDim.x + threadIdx.x;
    if (pid >= view.size) return;

    Particle p;
    p.readCoordinate(view.particles, pid);

    float sdf = sdfs[pid];

    if (sdf + h >= 0.0f)
    {
        float3 f = -gradients[pid] * min( maxForce, C * max(sdf + h, 0.0f) );
        atomicAdd(view.forces + pid, f);
    }
}
} // WallRepulsionPluginKernels

WallRepulsionPlugin::WallRepulsionPlugin(const YmrState *state, std::string name,
                                         std::string pvName, std::string wallName,
                                         float C, float h, float maxForce) :
    SimulationPlugin(state, name),
    pvName(pvName),
    wallName(wallName),
    C(C),
    h(h),
    maxForce(maxForce)
{}

void WallRepulsionPlugin::setup(Simulation* simulation, const MPI_Comm& comm, const MPI_Comm& interComm)
{
    SimulationPlugin::setup(simulation, comm, interComm);

    pv = simulation->getPVbyNameOrDie(pvName);
    wall = dynamic_cast<SDF_basedWall*>(simulation->getWallByNameOrDie(wallName));
    
    pv->requireDataPerParticle<float>(ChannelNames::sdf, ExtraDataManager::PersistenceMode::None);
    pv->requireDataPerParticle<float3>(ChannelNames::grad_sdf, ExtraDataManager::PersistenceMode::None);

    if (wall == nullptr)
        die("Wall repulsion plugin '%s' can only work with SDF-based walls, but got wall '%s'",
            name.c_str(), wallName.c_str());
}


// TODO: make that force be computed on halo also
// to get rid of the SDF wall margin
void WallRepulsionPlugin::beforeIntegration(hipStream_t stream)
{
    PVview view(pv, pv->local());
    
    auto sdfs      = pv->local()->extraPerParticle.getData<float>(ChannelNames::sdf);
    auto gradients = pv->local()->extraPerParticle.getData<float3>(ChannelNames::grad_sdf);

    float gradientThreshold = h + 0.1f;
    
    wall->sdfPerParticle(pv->local(), sdfs, gradients, gradientThreshold, stream);

    const int nthreads = 128;
    SAFE_KERNEL_LAUNCH(
            WallRepulsionPluginKernels::forceFromSDF,
            getNblocks(view.size, nthreads), nthreads, 0, stream,
            view, sdfs->devPtr(), gradients->devPtr(), C, h, maxForce );
}

