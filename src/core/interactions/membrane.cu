#include "hip/hip_runtime.h"
#include "membrane.h"

#include "membrane/common.h"

#include <core/pvs/membrane_vector.h>
#include <core/pvs/views/ov.h>
#include <core/utils/cuda_common.h>
#include <core/utils/kernel_launch.h>


namespace InteractionMembraneKernels
{
__global__ void computeAreaAndVolume(OVviewWithAreaVolume view, MeshView mesh)
{
    int objId = blockIdx.x;
    int offset = objId * mesh.nvertices;
    float2 a_v = make_float2(0.0f);

    for (int i = threadIdx.x; i < mesh.ntriangles; i += blockDim.x) {
        int3 ids = mesh.triangles[i];

        auto v0 = make_real3(f4tof3( view.particles[ 2 * (offset + ids.x) ] ));
        auto v1 = make_real3(f4tof3( view.particles[ 2 * (offset + ids.y) ] ));
        auto v2 = make_real3(f4tof3( view.particles[ 2 * (offset + ids.z) ] ));

        a_v.x += triangleArea(v0, v1, v2);
        a_v.y += triangleSignedVolume(v0, v1, v2);
    }

    a_v = warpReduce( a_v, [] (float a, float b) { return a+b; } );

    if (__laneid() == 0)
        atomicAdd(&view.area_volumes[objId], a_v);
}
} // namespace InteractionMembraneKernels

InteractionMembrane::InteractionMembrane(const YmrState *state, std::string name) :
    Interaction(state, name, /* default cutoff rc */ 1.0),
    impl(nullptr)
{}

InteractionMembrane::~InteractionMembrane() = default;

void InteractionMembrane::setPrerequisites(ParticleVector *pv1, ParticleVector *pv2, CellList *cl1, CellList *cl2)
{
    if (pv1 != pv2)
        die("Internal membrane forces can't be computed between two different particle vectors");

    auto ov = dynamic_cast<MembraneVector*>(pv1);
    if (ov == nullptr)
        die("Internal membrane forces can only be computed with a MembraneVector");

    ov->requireDataPerObject<float2>(ChannelNames::areaVolumes, ExtraDataManager::PersistenceMode::None);
}

void InteractionMembrane::local(ParticleVector *pv1, ParticleVector *pv2, CellList *cl1, CellList *cl2, hipStream_t stream)
{
    if (impl.get() == nullptr)
        die("%s needs a concrete implementation, none was provided", name.c_str());

    precomputeQuantities(pv1, stream);
    impl->local(pv1, pv2, cl1, cl2, stream);
}

void InteractionMembrane::halo(ParticleVector *pv1, ParticleVector *pv2, CellList *cl1, CellList *cl2, hipStream_t stream)
{
    debug("Not computing internal membrane forces between local and halo membranes of '%s'",
          pv1->name.c_str());
}
    
void InteractionMembrane::precomputeQuantities(ParticleVector *pv1, hipStream_t stream)
{
    auto ov = dynamic_cast<MembraneVector *>(pv1);

    if (ov->objSize != ov->mesh->getNvertices())
        die("Object size of '%s' (%d) and number of vertices (%d) mismatch",
            ov->name.c_str(), ov->objSize, ov->mesh->getNvertices());

    debug("Computing areas and volumes for %d cells of '%s'",
          ov->local()->nObjects, ov->name.c_str());

    OVviewWithAreaVolume view(ov, ov->local());

    MembraneMeshView mesh(static_cast<MembraneMesh*>(ov->mesh.get()));

    ov->local()
        ->extraPerObject.getData<float2>(ChannelNames::areaVolumes)
        ->clearDevice(stream);
    
    const int nthreads = 128;
    SAFE_KERNEL_LAUNCH(InteractionMembraneKernels::computeAreaAndVolume,
                       view.nObjects, nthreads, 0, stream,
                       view, mesh);
}
