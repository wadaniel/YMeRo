#include "dpd.h"
#include "pairwise.impl.h"
#include "pairwise_interactions/dpd.h"

#include <core/celllist.h>
#include <core/utils/make_unique.h>
#include <core/pvs/particle_vector.h>

#include <memory>

InteractionDPD::InteractionDPD(const YmrState *state, std::string name, float rc, float a, float gamma, float kbt, float power, bool allocateImpl) :
    Interaction(state, name, rc),
    a(a), gamma(gamma), kbt(kbt), power(power)
{
    if (allocateImpl) {
        PairwiseDPD dpd(rc, a, gamma, kbt, state->dt, power);
        impl = std::make_unique<InteractionPair<PairwiseDPD>> (state, name, rc, dpd);
    }
}

InteractionDPD::InteractionDPD(const YmrState *state, std::string name, float rc, float a, float gamma, float kbt, float power) :
    InteractionDPD(state, name, rc, a, gamma, kbt, power, true)
{}

InteractionDPD::~InteractionDPD() = default;

void InteractionDPD::setPrerequisites(ParticleVector *pv1, ParticleVector *pv2, CellList *cl1, CellList *cl2)
{
    impl->setPrerequisites(pv1, pv2, cl1, cl2);
}

std::vector<Interaction::InteractionChannel> InteractionDPD::getFinalOutputChannels() const
{
    return impl->getFinalOutputChannels();
}

void InteractionDPD::local(ParticleVector *pv1, ParticleVector *pv2,
                           CellList *cl1, CellList *cl2,
                           hipStream_t stream)
{
    impl->local(pv1, pv2, cl1, cl2, stream);
}

void InteractionDPD::halo(ParticleVector *pv1, ParticleVector *pv2,
                          CellList *cl1, CellList *cl2,
                          hipStream_t stream)
{
    impl->halo(pv1, pv2, cl1, cl2, stream);
}

void InteractionDPD::setSpecificPair(ParticleVector *pv1, ParticleVector *pv2, 
                                     float a, float gamma, float kbt, float power)
{
    if (a     == Default) a     = this->a;
    if (gamma == Default) gamma = this->gamma;
    if (kbt   == Default) kbt   = this->kbt;
    if (power == Default) power = this->power;

    PairwiseDPD dpd(this->rc, a, gamma, kbt, state->dt, power);
    auto ptr = static_cast< InteractionPair<PairwiseDPD>* >(impl.get());
    
    ptr->setSpecificPair(pv1->name, pv2->name, dpd);
}


