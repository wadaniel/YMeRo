#include "hip/hip_runtime.h"
#include "impose_velocity.h"

#include <core/pvs/particle_vector.h>
#include <core/pvs/views/pv.h>
#include <core/simulation.h>
#include <core/utils/cuda_common.h>
#include <core/utils/cuda_rng.h>
#include <core/utils/kernel_launch.h>

namespace ImposeVelocityKernels
{
__global__ void addVelocity(PVview view, DomainInfo domain, float3 low, float3 high, float3 extraVel)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid >= view.size) return;

    Particle p(view.particles, gid);
    float3 gr = domain.local2global(p.r);

    if (low.x <= gr.x && gr.x <= high.x &&
        low.y <= gr.y && gr.y <= high.y &&
        low.z <= gr.z && gr.z <= high.z)
    {
        p.u += extraVel;
        view.particles[2*gid+1] = p.u2Float4();
    }
}

__global__ void averageVelocity(PVview view, DomainInfo domain, float3 low, float3 high, double3* totVel, int* nSamples)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    Particle p;

    p.u = make_float3(0.f);

    if (gid < view.size) {

        p.read(view.particles, gid);
        float3 gr = domain.local2global(p.r);

        if (low.x <= gr.x && gr.x <= high.x &&
            low.y <= gr.y && gr.y <= high.y &&
            low.z <= gr.z && gr.z <= high.z)
        {
            atomicAggInc(nSamples);
        }
        else
        {
            p.u = make_float3(0.0f);
        }
    }
    
    float3 u = warpReduce(p.u, [](float a, float b) { return a+b; });
    if (__laneid() == 0 && dot(u, u) > 1e-8f)
    {
        atomicAdd(&totVel[0].x, (double)u.x);
        atomicAdd(&totVel[0].y, (double)u.y);
        atomicAdd(&totVel[0].z, (double)u.z);
    }
}
} // namespace ImposeVelocityKernels

ImposeVelocityPlugin::ImposeVelocityPlugin(const YmrState *state, std::string name, std::vector<std::string> pvNames,
                                           float3 low, float3 high, float3 targetVel, int every) :
    SimulationPlugin(state, name),
    pvNames(pvNames),
    low(low),
    high(high),
    targetVel(targetVel),
    every(every)
{}

void ImposeVelocityPlugin::setup(Simulation* simulation, const MPI_Comm& comm, const MPI_Comm& interComm)
{
    SimulationPlugin::setup(simulation, comm, interComm);

    for (auto& nm : pvNames)
        pvs.push_back(simulation->getPVbyNameOrDie(nm));
}

void ImposeVelocityPlugin::afterIntegration(hipStream_t stream)
{
    if (state->currentStep % every == 0)
    {
        const int nthreads = 128;

        totVel.clearDevice(stream);
        nSamples.clearDevice(stream);
        
        for (auto& pv : pvs)
            SAFE_KERNEL_LAUNCH(
                    ImposeVelocityKernels::averageVelocity,
                    getNblocks(pv->local()->size(), nthreads), nthreads, 0, stream,
                    PVview(pv, pv->local()), state->domain, low, high, totVel.devPtr(), nSamples.devPtr() );

        totVel.downloadFromDevice(stream, ContainersSynch::Asynch);
        nSamples.downloadFromDevice(stream);

        float3 avgVel = make_float3(totVel[0].x / nSamples[0], totVel[0].y / nSamples[0], totVel[0].z / nSamples[0]);

        debug("Current mean velocity measured by plugin '%s' is [%f %f %f]; as of %d particles",
              name.c_str(), avgVel.x, avgVel.y, avgVel.z, nSamples[0]);

        for (auto& pv : pvs)
            SAFE_KERNEL_LAUNCH(
                    ImposeVelocityKernels::addVelocity,
                    getNblocks(pv->local()->size(), nthreads), nthreads, 0, stream,
                    PVview(pv, pv->local()), state->domain, low, high, targetVel - avgVel);
    }
}

void ImposeVelocityPlugin::setTargetVelocity(PyTypes::float3 v)
{
    info("Changing target velocity from [%f %f %f] to [%f %f %f]",
         targetVel.x, targetVel.y, targetVel.z,
         std::get<0>(v), std::get<1>(v), std::get<2>(v));
    
    targetVel = make_float3(v);
}

