#include "hip/hip_runtime.h"
#include "average_relative_flow.h"

#include "utils/sampling_helpers.h"
#include "utils/simple_serializer.h"

#include <core/celllist.h>
#include <core/pvs/object_vector.h>
#include <core/pvs/particle_vector.h>
#include <core/pvs/views/pv.h>
#include <core/rigid_kernels/rigid_motion.h>
#include <core/simulation.h>
#include <core/utils/cuda_common.h>
#include <core/utils/kernel_launch.h>

namespace AverageRelativeFlowKernels
{
__global__ void sampleRelative(
        PVview pvView, CellListInfo cinfo,
        float* avgDensity,
        ChannelsInfo channelsInfo,
        float3 relativePoint)
{
    const int pid = threadIdx.x + blockIdx.x*blockDim.x;
    if (pid >= pvView.size) return;

    Particle p(pvView.particles, pid);
    p.r -= relativePoint;

    int3 cid3 = cinfo.getCellIdAlongAxes<CellListsProjection::NoClamp>(p.r);
    cid3 = (cid3 + cinfo.ncells) % cinfo.ncells;
    const int cid = cinfo.encode(cid3);

    atomicAdd(avgDensity + cid, 1);

    SamplingHelpersKernels::sampleChannels(pid, cid, channelsInfo);
}
} // namespace AverageRelativeFlowKernels

AverageRelative3D::AverageRelative3D(
    const YmrState *state, std::string name, std::vector<std::string> pvNames,
    std::vector<std::string> channelNames,
    std::vector<Average3D::ChannelType> channelTypes, int sampleEvery,
    int dumpEvery, float3 binSize, std::string relativeOVname, int relativeID) :
    Average3D(state, name, pvNames, channelNames, channelTypes, sampleEvery,
              dumpEvery, binSize),
    relativeOVname(relativeOVname), relativeID(relativeID)
{}

void AverageRelative3D::setup(Simulation* simulation, const MPI_Comm& comm, const MPI_Comm& interComm)
{
    Average3D::setup(simulation, comm, interComm);

    int local_size = density.size();
    int global_size = local_size * nranks;
    
    localDensity.resize(local_size);
    density.resize_anew(global_size);
    accumulated_density.resize_anew(global_size);
    density.clear(0);

    localChannels.resize(channelsInfo.n);

    for (int i = 0; i < channelsInfo.n; i++) {
        local_size = channelsInfo.average[i].size();
        global_size = local_size * nranks;
        localChannels[i].resize(local_size);
        channelsInfo.average[i].resize_anew(global_size);
        accumulated_average [i].resize_anew(global_size);
        channelsInfo.average[i].clear(0);
        channelsInfo.averagePtrs[i] = channelsInfo.average[i].devPtr();
    }

    channelsInfo.averagePtrs.uploadToDevice(0);
    channelsInfo.types.uploadToDevice(0);

    // Relative stuff
    relativeOV = simulation->getOVbyNameOrDie(relativeOVname);

    if ( !relativeOV->local()->extraPerObject.checkChannelExists(ChannelNames::motions) )
        die("Only rigid objects are supported for relative flow, but got OV '%s'", relativeOV->name.c_str());

    int locsize = relativeOV->local()->nObjects;
    int totsize;

    MPI_Check( MPI_Reduce(&locsize, &totsize, 1, MPI_INT, MPI_SUM, 0, comm) );

    if (rank == 0 && relativeID >= totsize)
        die("Too few objects in OV '%s' (only %d); but requested id %d",
            relativeOV->name.c_str(), totsize, relativeID);
}

void AverageRelative3D::sampleOnePv(float3 relativeParam, ParticleVector *pv, hipStream_t stream)
{
    CellListInfo cinfo(binSize, state->domain.globalSize);
    PVview pvView(pv, pv->local());
    ChannelsInfo gpuInfo(channelsInfo, pv, stream);

    const int nthreads = 128;
    SAFE_KERNEL_LAUNCH
        (AverageRelativeFlowKernels::sampleRelative,
         getNblocks(pvView.size, nthreads), nthreads, 0, stream,
         pvView, cinfo, density.devPtr(), gpuInfo, relativeParam);
}

void AverageRelative3D::afterIntegration(hipStream_t stream)
{
    const int TAG = 22;
    const int NCOMPONENTS = 2 * sizeof(float3) / sizeof(float);
    
    if (state->currentStep % sampleEvery != 0 || state->currentStep == 0) return;

    debug2("Plugin %s is sampling now", name.c_str());

    float3 relativeParams[2] = {make_float3(0.0f), make_float3(0.0f)};

    // Find and broadcast the position and velocity of the relative object
    MPI_Request req;
    MPI_Check( MPI_Irecv(relativeParams, NCOMPONENTS, MPI_FLOAT, MPI_ANY_SOURCE, TAG, comm, &req) );

    auto ids     = relativeOV->local()->extraPerObject.getData<int>(ChannelNames::globalIds);
    auto motions = relativeOV->local()->extraPerObject.getData<RigidMotion>(ChannelNames::motions);

    ids    ->downloadFromDevice(stream, ContainersSynch::Asynch);
    motions->downloadFromDevice(stream, ContainersSynch::Synch);

    for (int i = 0; i < ids->size(); i++)
    {
        if ((*ids)[i] == relativeID)
        {
            float3 params[2] = { make_float3( (*motions)[i].r   ),
                                 make_float3( (*motions)[i].vel ) };

            params[0] = state->domain.local2global(params[0]);

            for (int r = 0; r < nranks; r++)
                MPI_Send(&params, NCOMPONENTS, MPI_FLOAT, r, TAG, comm);

            break;
        }
    }

    MPI_Check( MPI_Wait(&req, MPI_STATUS_IGNORE) );

    relativeParams[0] = state->domain.global2local(relativeParams[0]);

    for (auto& pv : pvs) sampleOnePv(relativeParams[0], pv, stream);

    accumulateSampledAndClear(stream);
    
    averageRelativeVelocity += relativeParams[1];

    nSamples++;
}


void AverageRelative3D::extractLocalBlock()
{
    static const double scale_by_density = -1.0;
    
    auto oneChannel = [this] (const PinnedBuffer<double>& channel, Average3D::ChannelType type, double scale, std::vector<double>& dest) {

        MPI_Check( MPI_Allreduce(MPI_IN_PLACE, channel.hostPtr(), channel.size(), MPI_DOUBLE, MPI_SUM, comm) );

        int ncomponents = this->getNcomponents(type);

        int3 globalResolution = resolution * nranks3D;

        double factor;
        int dstId = 0;
        for (int k = rank3D.z*resolution.z; k < (rank3D.z+1)*resolution.z; k++) {
            for (int j = rank3D.y*resolution.y; j < (rank3D.y+1)*resolution.y; j++) {
                for (int i = rank3D.x*resolution.x; i < (rank3D.x+1)*resolution.x; i++) {                    
                    int scalId = (k*globalResolution.y*globalResolution.x + j*globalResolution.x + i);
                    int srcId = ncomponents * scalId;
                    for (int c = 0; c < ncomponents; c++) {
                        if (scale == scale_by_density) factor = 1.0f / accumulated_density[scalId];
                        else                           factor = scale;

                        dest[dstId++] = channel[srcId] * factor;
                        srcId++;
                    }
                }
            }
        }
    };

    // Order is important! Density comes first
    oneChannel(accumulated_density, Average3D::ChannelType::Scalar, 1.0 / (nSamples * binSize.x*binSize.y*binSize.z), localDensity);

    for (int i = 0; i < channelsInfo.n; i++)
        oneChannel(accumulated_average[i], channelsInfo.types[i], scale_by_density, localChannels[i]);
}

void AverageRelative3D::serializeAndSend(hipStream_t stream)
{
    if (state->currentStep % dumpEvery != 0 || state->currentStep == 0) return;

    for (int i = 0; i < channelsInfo.n; i++) {
        auto& data = accumulated_average[i];

        if (channelsInfo.names[i] == "velocity") {
            const int nthreads = 128;

            SAFE_KERNEL_LAUNCH
                (SamplingHelpersKernels::correctVelocity,
                 getNblocks(data.size() / 3, nthreads), nthreads, 0, stream,
                 data.size() / 3, (double3*)data.devPtr(), accumulated_density.devPtr(), averageRelativeVelocity / (float) nSamples);

            averageRelativeVelocity = make_float3(0);
        }
    }

        
    accumulated_density.downloadFromDevice(stream, ContainersSynch::Asynch);
    accumulated_density.clearDevice(stream);
    
    for (auto& data : accumulated_average)
    {
        data.downloadFromDevice(stream, ContainersSynch::Asynch);
        data.clearDevice(stream);
    }

    CUDA_Check( hipStreamSynchronize(stream) );

    extractLocalBlock();
    nSamples = 0;


    debug2("Plugin '%s' is now packing the data", name.c_str());
    waitPrevSend();
    SimpleSerializer::serialize(sendBuffer, state->currentTime, localDensity, localChannels);
    send(sendBuffer);
}

