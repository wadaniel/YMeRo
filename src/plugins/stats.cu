#include "hip/hip_runtime.h"
#include "stats.h"
#include "utils/simple_serializer.h"

#include <core/datatypes.h>
#include <core/pvs/particle_vector.h>
#include <core/pvs/views/pv.h>
#include <core/simulation.h>
#include <core/utils/cuda_common.h>
#include <core/utils/kernel_launch.h>

namespace StatsKernels
{
using Stats::ReductionType;

__global__ void totalMomentumEnergy(PVview view, ReductionType *momentum, ReductionType *energy, float* maxvel)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    float3 vel, myMomentum;
    float myEnergy = 0.f, myMaxIvelI;
    vel = myMomentum = make_float3(0.f);

    if (tid < view.size)
    {
        vel        = make_float3(view.particles[2*tid+1]);
        myMomentum = vel * view.mass;
        myEnergy   = dot(vel, vel) * view.mass * 0.5f;
    }
    
    myMomentum = warpReduce(myMomentum, [](float a, float b) { return a+b; });
    myEnergy   = warpReduce(myEnergy,   [](float a, float b) { return a+b; });
    
    myMaxIvelI = warpReduce(length(vel), [](float a, float b) { return max(a, b); });

    if (__laneid() == 0)
    {
        atomicAdd(momentum+0, (ReductionType)myMomentum.x);
        atomicAdd(momentum+1, (ReductionType)myMomentum.y);
        atomicAdd(momentum+2, (ReductionType)myMomentum.z);
        atomicAdd(energy,     (ReductionType)myEnergy);

        atomicMax((int*)maxvel, __float_as_int(myMaxIvelI));
    }
}
} // namespace StatsKernels
    
SimulationStats::SimulationStats(const YmrState *state, std::string name, int fetchEvery) :
    SimulationPlugin(state, name),
    fetchEvery(fetchEvery)
{
    timer.start();
}

SimulationStats::~SimulationStats() = default;

void SimulationStats::setup(Simulation *simulation, const MPI_Comm& comm, const MPI_Comm& interComm)
{
    SimulationPlugin::setup(simulation, comm, interComm);
    pvs = simulation->getParticleVectors();
}

void SimulationStats::afterIntegration(hipStream_t stream)
{
    if (state->currentStep % fetchEvery != 0) return;

    momentum.clear(stream);
    energy  .clear(stream);
    maxvel  .clear(stream);

    nparticles = 0;
    for (auto& pv : pvs)
    {
        PVview view(pv, pv->local());

        SAFE_KERNEL_LAUNCH(
                StatsKernels::totalMomentumEnergy,
                getNblocks(view.size, 128), 128, 0, stream,
                view, momentum.devPtr(), energy.devPtr(), maxvel.devPtr() );

        nparticles += view.size;
    }

    momentum.downloadFromDevice(stream, ContainersSynch::Asynch);
    energy  .downloadFromDevice(stream, ContainersSynch::Asynch);
    maxvel  .downloadFromDevice(stream);

    needToDump = true;
}

void SimulationStats::serializeAndSend(hipStream_t stream)
{
    if (needToDump)
    {
        float tm = timer.elapsedAndReset() / (state->currentStep < fetchEvery ? 1.0f : fetchEvery);
        waitPrevSend();
        SimpleSerializer::serialize(sendBuffer, tm, state->currentTime, state->currentStep, nparticles, momentum, energy, maxvel);
        send(sendBuffer);
        needToDump = false;
    }
}

PostprocessStats::PostprocessStats(std::string name, std::string filename) :
        PostprocessPlugin(name)
{
    if (std::is_same<Stats::ReductionType, float>::value)
        mpiReductionType = MPI_FLOAT;
    else if (std::is_same<Stats::ReductionType, double>::value)
        mpiReductionType = MPI_DOUBLE;
    else
        die("Incompatible type");

    if (filename != "")
    {
        fdump = fopen(filename.c_str(), "w");
        if (!fdump) die("Could not open file '%s'", filename.c_str());
        fprintf(fdump, "# time  kBT  vx vy vz  max(abs(v))  simulation_time_per_step(ms)\n");
    }
}

PostprocessStats::~PostprocessStats()
{
    if (fdump != nullptr) fclose(fdump);
}

void PostprocessStats::deserialize(MPI_Status& stat)
{
    TimeType currentTime;
    float realTime;
    int nparticles, currentTimeStep;
    int maxNparticles, minNparticles;

    std::vector<Stats::ReductionType> momentum, energy;
    std::vector<float> maxvel;

    SimpleSerializer::deserialize(data, realTime, currentTime, currentTimeStep, nparticles, momentum, energy, maxvel);

    MPI_Check( MPI_Reduce(&nparticles, &minNparticles, 1, MPI_INT, MPI_MIN, 0, comm) );
    MPI_Check( MPI_Reduce(&nparticles, &maxNparticles, 1, MPI_INT, MPI_MAX, 0, comm) );
    
    MPI_Check( MPI_Reduce(rank == 0 ? MPI_IN_PLACE : &nparticles,     &nparticles,     1, MPI_INT,          MPI_SUM, 0, comm) );
    MPI_Check( MPI_Reduce(rank == 0 ? MPI_IN_PLACE : energy.data(),   energy.data(),   1, mpiReductionType, MPI_SUM, 0, comm) );
    MPI_Check( MPI_Reduce(rank == 0 ? MPI_IN_PLACE : momentum.data(), momentum.data(), 3, mpiReductionType, MPI_SUM, 0, comm) );

    MPI_Check( MPI_Reduce(rank == 0 ? MPI_IN_PLACE : maxvel.data(),   maxvel.data(),   1, MPI_FLOAT,        MPI_MAX, 0, comm) );

    MPI_Check( MPI_Reduce(rank == 0 ? MPI_IN_PLACE : &realTime,       &realTime,       1, MPI_FLOAT,        MPI_MAX, 0, comm) );

    if (rank == 0)
    {
        momentum[0] /= (double)nparticles;
        momentum[1] /= (double)nparticles;
        momentum[2] /= (double)nparticles;
        const Stats::ReductionType temperature = energy[0] / ( (3/2.0)*nparticles );

        printf("Stats at timestep %d (simulation time %f):\n", currentTimeStep, currentTime);
        printf("\tOne timestep takes %.2f ms", realTime);
        printf("\tNumber of particles (total, min/proc, max/proc): %d,  %d,  %d\n", nparticles, minNparticles, maxNparticles);
        printf("\tAverage momentum: [%e %e %e]\n", momentum[0], momentum[1], momentum[2]);
        printf("\tMax velocity magnitude: %f\n", maxvel[0]);
        printf("\tTemperature: %.4f\n\n", temperature);

        if (fdump != nullptr)
        {
            fprintf(fdump, "%g %g %g %g %g %g %g\n", currentTime,
                    temperature, momentum[0], momentum[1], momentum[2], maxvel[0], realTime);
            fflush(fdump);
        }
    }
}


