#include "hip/hip_runtime.h"
#include <core/pvs/particle_vector.h>
#include <core/pvs/views/pv.h>
#include <core/pvs/object_vector.h>
#include <core/celllist.h>
#include <core/utils/cuda_common.h>
#include <core/utils/kernel_launch.h>
#include <core/utils/typeMap.h>
#include <core/logger.h>

#include <extern/cub/cub/device/device_scan.cuh>

namespace CellListKernels
{

enum {INVALID = -1};

inline __device__ bool outgoingParticle(float4 pos)
{
    return Float3_int(pos).isMarked();
}

__global__ void computeCellSizes(PVview view, CellListInfo cinfo)
{
    const int pid = blockIdx.x * blockDim.x + threadIdx.x;
    if (pid >= view.size) return;

    float4 coo = readNoCache(view.particles + pid*2);
    int cid = cinfo.getCellId(coo);

    // XXX: relying here only on redistribution
    if ( !outgoingParticle(coo) )
        atomicAdd(cinfo.cellSizes + cid, 1);
}

__global__ void reorderParticles(PVview view, CellListInfo cinfo, float4 *outParticles)
{
    const int gid = blockIdx.x * blockDim.x + threadIdx.x;
    const int pid = gid / 2;
    const int sh  = gid % 2;  // sh = 0 copies coordinates, sh = 1 -- velocity
    if (pid >= view.size) return;

    int dstId;

    // this is to allow more cache for atomics
    // loads / stores here need no cache
    float4 val = readNoCache(view.particles+gid);

    int cid;
    if (sh == 0)
    {
        cid = cinfo.getCellId(val);

        //  XXX: relying here only on redistribution
        if ( !outgoingParticle(val) )
            dstId = cinfo.cellStarts[cid] + atomicAdd(cinfo.cellSizes + cid, 1);
        else
            dstId = -1;
    }

    int otherDst = warpShflUp(dstId, 1);
    if (sh == 1)
        dstId = otherDst;

    if (dstId >= 0)
    {
        writeNoCache(outParticles + 2*dstId+sh, val);
        if (sh == 0) cinfo.order[pid] = dstId;
    }
    else if (sh == 0)
        cinfo.order[pid] = INVALID;
}

template <typename T>
__global__ void reorderExtraDataPerParticle(int n, const T *inExtraData, CellListInfo cinfo, T *outExtraData)
{
    int srcId = blockIdx.x * blockDim.x + threadIdx.x;
    if (srcId >= n) return;

    int dstId = cinfo.order[srcId];
    if (dstId != INVALID)
        outExtraData[dstId] = inExtraData[srcId];
}

__global__ void addForcesKernel(PVview dstView, CellListInfo cinfo, PVview srcView)
{
    int pid = blockIdx.x * blockDim.x + threadIdx.x;
    if (pid >= dstView.size) return;

    int srcId = cinfo.order[pid];

    assert(srcId != INVALID);
    dstView.forces[pid] += srcView.forces[srcId];
}

template <typename T>
__global__ void accumulateKernel(int n, T *dst, CellListInfo cinfo, const T *src)
{
    int pid = blockIdx.x * blockDim.x + threadIdx.x;
    if (pid >= n) return;

    int srcId = cinfo.order[pid];

    assert(srcId != INVALID);
    dst[pid] += src[srcId];
}

} // namespace CellListKernels

//=================================================================================
// Info
//=================================================================================

CellListInfo::CellListInfo(float rc, float3 localDomainSize) :
        rc(rc), h(make_float3(rc)), localDomainSize(localDomainSize)
{
    ncells = make_int3( floorf(localDomainSize / rc + 1e-6) );
    float3 h = make_float3(localDomainSize) / make_float3(ncells);
    invh = 1.0f / h;
    this->rc = std::min( {h.x, h.y, h.z} );

    totcells = ncells.x * ncells.y * ncells.z;
}

CellListInfo::CellListInfo(float3 h, float3 localDomainSize) :
        h(h), invh(1.0f/h), localDomainSize(localDomainSize)
{
    rc = std::min( {h.x, h.y, h.z} );
    ncells = make_int3( ceilf(localDomainSize / h - 1e-6f) );
    totcells = ncells.x * ncells.y * ncells.z;
}

//=================================================================================
// Basic cell-lists
//=================================================================================

CellList::CellList(ParticleVector *pv, float rc, float3 localDomainSize) :
        CellListInfo(rc, localDomainSize), pv(pv),
        particlesDataContainer(new LocalParticleVector(nullptr))
{
    localPV = particlesDataContainer.get();
    
    cellSizes. resize_anew(totcells + 1);
    cellStarts.resize_anew(totcells + 1);

    cellSizes. clear(0);
    cellStarts.clear(0);
    CUDA_Check( hipStreamSynchronize(0) );

    debug("Initialized %s cell-list with %dx%dx%d cells and cut-off %f", pv->name.c_str(), ncells.x, ncells.y, ncells.z, this->rc);
}

CellList::CellList(ParticleVector *pv, int3 resolution, float3 localDomainSize) :
        CellListInfo(localDomainSize / make_float3(resolution), localDomainSize), pv(pv),
        particlesDataContainer(new LocalParticleVector(nullptr))
{
    localPV = particlesDataContainer.get();
    
    cellSizes. resize_anew(totcells + 1);
    cellStarts.resize_anew(totcells + 1);

    cellSizes. clear(0);
    cellStarts.clear(0);
    CUDA_Check( hipStreamSynchronize(0) );

    debug("Initialized %s cell-list with %dx%dx%d cells and cut-off %f", pv->name.c_str(), ncells.x, ncells.y, ncells.z, this->rc);
}

CellList::~CellList() = default;

bool CellList::_checkNeedBuild() const
{
    if (changedStamp == pv->cellListStamp)
    {
        debug2("%s is already up-to-date, building skipped", makeName().c_str());
        return false;
    }

    if (pv->local()->size() == 0)
    {
        debug2("%s consists of no particles, building skipped", makeName().c_str());
        return false;
    }

    return true;
}

template <typename T>
static void requireData(const std::string& channelName, int np, ExtraDataManager& containerManager)
{
    if (!containerManager.checkChannelExists(channelName))
        containerManager.createData<T>(channelName, np);
}


void CellList::_updateExtraDataChannels(hipStream_t stream)
{
    auto& pvManager        = pv->local()->extraPerParticle;
    auto& containerManager = particlesDataContainer->extraPerParticle;
    int np = pv->local()->size();

    for (const auto& namedChannel : pvManager.getSortedChannels()) {
        const auto& name = namedChannel.first;
        const auto& desc = namedChannel.second;
        if (desc->persistence != ExtraDataManager::PersistenceMode::Persistent) continue;

#define SWITCH_ENTRY(ctype)                                             \
        case DataType::TOKENIZE(ctype):                                 \
            requireData<ctype>(name, np, containerManager);             \
            break;

        switch(desc->dataType) {
            TYPE_TABLE(SWITCH_ENTRY);
        default:
            die("%s: cannot require extra data: %s has None type.", makeName().c_str(), name.c_str());
        }

#undef SWITCH_ENTRY        
    }
}

void CellList::_computeCellSizes(hipStream_t stream)
{
    debug2("%s : Computing cell sizes for %d particles", makeName().c_str(), pv->local()->size());
    cellSizes.clear(stream);

    PVview view(pv, pv->local());

    const int nthreads = 128;
    SAFE_KERNEL_LAUNCH(
            CellListKernels::computeCellSizes,
            getNblocks(view.size, nthreads), nthreads, 0, stream,
            view, cellInfo() );
}

void CellList::_computeCellStarts(hipStream_t stream)
{
	// Scan is always working with the same number of cells
	// Memory requirements can't change
	size_t bufSize = scanBuffer.size();

	if (bufSize == 0)
	{
		hipcub::DeviceScan::ExclusiveSum(nullptr, bufSize, cellSizes.devPtr(), cellStarts.devPtr(), totcells+1, stream);
		scanBuffer.resize_anew(bufSize);
	}
    hipcub::DeviceScan::ExclusiveSum(scanBuffer.devPtr(), bufSize,
    							  cellSizes.devPtr(), cellStarts.devPtr(), totcells+1, stream);
}

void CellList::_reorderData(hipStream_t stream)
{
    debug2("Reordering %d %s particles", pv->local()->size(), pv->name.c_str());

    PVview view(pv, pv->local());

    order.resize_anew(view.size);
    particlesDataContainer->resize_anew(view.size);
    cellSizes.clear(stream);

    const int nthreads = 128;
    SAFE_KERNEL_LAUNCH(
        CellListKernels::reorderParticles,
        getNblocks(2*view.size, nthreads), nthreads, 0, stream,
        view, cellInfo(), (float4*)particlesDataContainer->coosvels.devPtr() );
}

template <typename T>
static void reorderExtraDataEntry(int np, CellListInfo cinfo, ExtraDataManager *dstExtraData,
                                  const ExtraDataManager::ChannelDescription *channel, const std::string& channelName,
                                  hipStream_t stream)
{
    T      *outExtraData = dstExtraData->getData<T>(channelName)->devPtr();
    const T *inExtraData = (const T*) channel->container->genericDevPtr();

    const int nthreads = 128;

    SAFE_KERNEL_LAUNCH(
        CellListKernels::reorderExtraDataPerParticle<T>,
        getNblocks(np, nthreads), nthreads, 0, stream,
        np, inExtraData, cinfo, outExtraData );
}

void CellList::_reorderExtraDataEntry(const std::string& channelName,
                                      const ExtraDataManager::ChannelDescription *channelDesc,
                                      hipStream_t stream)
{
    auto dstExtraData = &particlesDataContainer->extraPerParticle;
    int np = pv->local()->size();
    
    switch (channelDesc->dataType)
    {

#define SWITCH_ENTRY(ctype)                             \
        case DataType::TOKENIZE(ctype):                 \
            reorderExtraDataEntry<ctype>                \
                (np, cellInfo(), dstExtraData,          \
                 channelDesc, channelName, stream);     \
            break;

        TYPE_TABLE(SWITCH_ENTRY);

#undef SWITCH_ENTRY

    default:
        die("%s : cannot reorder data: channel '%s' has None type",
            makeName().c_str(), channelName.c_str());
    };

}

void CellList::_reorderPersistentData(hipStream_t stream)
{
    auto srcExtraData = &pv->local()->extraPerParticle;
    
    for (const auto& namedChannel : srcExtraData->getSortedChannels()) {
        const auto& name = namedChannel.first;
        const auto& desc = namedChannel.second;
        if (desc->persistence != ExtraDataManager::PersistenceMode::Persistent) continue;
        _reorderExtraDataEntry(name, desc, stream);
    }
}

void CellList::_build(hipStream_t stream)
{
    _computeCellSizes(stream);
    _computeCellStarts(stream);
    _reorderData(stream);
    _reorderPersistentData(stream);
    
    changedStamp = pv->cellListStamp;
}

CellListInfo CellList::cellInfo()
{
    CellListInfo::cellSizes  = cellSizes.devPtr();
    CellListInfo::cellStarts = cellStarts.devPtr();
    CellListInfo::order      = order.devPtr();

    return *((CellListInfo*)this);
}

void CellList::build(hipStream_t stream)
{
    _updateExtraDataChannels(stream);
        
    if (!_checkNeedBuild()) return;
    
    debug("building %s", makeName().c_str());
    
    _build(stream);
}

void CellList::_accumulateForces(hipStream_t stream)
{
    PVview dstView(pv, pv->local());
    int nthreads = 128;

    SAFE_KERNEL_LAUNCH(
            CellListKernels::addForcesKernel,
            getNblocks(dstView.size, nthreads), nthreads, 0, stream,
            dstView, cellInfo(), getView<PVview>() );
}

void CellList::_accumulateExtraData(const std::string& channelName, hipStream_t stream)
{
    const int nthreads = 128;
    switch(localPV->extraPerParticle.getChannelDescOrDie(channelName).dataType) {

#define SWITCH_ENTRY(ctype)                                             \
        case DataType::TOKENIZE(ctype):                                 \
        {                                                               \
            auto src = localPV    ->extraPerParticle.getData<ctype>(channelName); \
            auto dst = pv->local()->extraPerParticle.getData<ctype>(channelName); \
            int n = pv->local()->size();                                \
            SAFE_KERNEL_LAUNCH(                                         \
                CellListKernels::accumulateKernel<ctype>,               \
                getNblocks(n, nthreads), nthreads, 0, stream,           \
                n, dst->devPtr(), cellInfo(), src->devPtr() );          \
            }                                                           \
            break;

        TYPE_TABLE_ADDITIONABLE(SWITCH_ENTRY);

    default:
        die("%s : cannot accumulate entry '%s': type not supported",
            makeName().c_str(), channelName.c_str());

#undef SWITCH_ENTRY
    };        
}

void CellList::accumulateChannels(const std::vector<std::string>& channelNames, hipStream_t stream)
{
    for (const auto& channelName : channelNames) {
        debug2("%s : accumulating channel '%s'", makeName().c_str(), channelName.c_str());

        if (channelName == ChannelNames::forces)
            _accumulateForces(stream);
        else
            _accumulateExtraData(channelName, stream);
    }
}

void CellList::gatherChannels(const std::vector<std::string>& channelNames, hipStream_t stream)
{
    for (auto& channelName : channelNames) {

        debug("%s : gathering channel '%s'", makeName().c_str(), channelName.c_str());
        
        auto& desc = localPV->extraPerParticle.getChannelDescOrDie(channelName);
        _reorderExtraDataEntry(channelName, &desc, stream);

        // invalidate particle vector halo if any entry is active
        pv->haloValid = false;
    }
}

void CellList::clearChannels(const std::vector<std::string>& channelNames, hipStream_t stream)
{
    for (const auto& channelName : channelNames) {
        debug2("%s : clearing channel '%s'", makeName().c_str(), channelName.c_str());

        if (channelName == ChannelNames::forces)
            localPV->forces.clear(stream);
        else
            localPV->extraPerParticle.getGenericData(channelName)->clearDevice(stream);
    }
}

LocalParticleVector* CellList::getLocalParticleVector() {return localPV;}

std::string CellList::makeName() const
{
    return "Cell List '" + pv->name + "' (rc " + std::to_string(rc) + ")";
}


//=================================================================================
// Primary cell-lists
//=================================================================================

PrimaryCellList::PrimaryCellList(ParticleVector *pv, float rc, float3 localDomainSize) :
        CellList(pv, rc, localDomainSize)
{
    localPV = pv->local();

    if (dynamic_cast<ObjectVector*>(pv) != nullptr)
        error("Using primary cell-lists with objects is STRONGLY discouraged. This will very likely result in an error");
}

PrimaryCellList::PrimaryCellList(ParticleVector *pv, int3 resolution, float3 localDomainSize) :
        CellList(pv, resolution, localDomainSize)
{
    localPV = pv->local();

    if (dynamic_cast<ObjectVector*>(pv) != nullptr)
        error("Using primary cell-lists with objects is STRONGLY discouraged. This will very likely result in an error");
}

PrimaryCellList::~PrimaryCellList() = default;

void PrimaryCellList::build(hipStream_t stream)
{
	// Reqired here to avoid ptr swap if building didn't actually happen
    if (!_checkNeedBuild()) return;

    CellList::build(stream);

    if (pv->local()->size() == 0)
    {
        debug2("%s consists of no particles, cell-list building skipped", pv->name.c_str());
        return;
    }
    
    // Now we need the new size of particles array.
    int newSize;
    CUDA_Check( hipMemcpyAsync(&newSize, cellStarts.devPtr() + totcells, sizeof(int), hipMemcpyDeviceToHost, stream) );
    CUDA_Check( hipStreamSynchronize(stream) );

    debug2("%s : reordering completed, new size of %s particle vector is %d",
           makeName().c_str(), pv->name.c_str(), newSize);

    particlesDataContainer->resize(newSize, stream);

    std::swap(pv->local()->coosvels, particlesDataContainer->coosvels);
    _swapPersistentExtraData();
    
    pv->local()->resize(newSize, stream);
}

void PrimaryCellList::accumulateChannels(const std::vector<std::string>& channelNames, hipStream_t stream)
{}

void PrimaryCellList::gatherChannels(const std::vector<std::string>& channelNames, hipStream_t stream)
{
    // do not need to reorder data, but still invalidate halo
    if (!channelNames.empty())
        pv->haloValid = false;
}


template <typename T>
static void swap(const std::string& channelName, ExtraDataManager& pvManager, ExtraDataManager& containerManager)
{
    std::swap(*pvManager       .getData<T>(channelName),
              *containerManager.getData<T>(channelName));
}

void PrimaryCellList::_swapPersistentExtraData()
{
    auto& pvManager        = pv->local()->extraPerParticle;
    auto& containerManager = particlesDataContainer->extraPerParticle;
    
    for (const auto& namedChannel : pvManager.getSortedChannels()) {
        const auto& name = namedChannel.first;
        const auto& desc = namedChannel.second;
        if (desc->persistence != ExtraDataManager::PersistenceMode::Persistent) continue;

#define SWITCH_ENTRY(ctype)                                             \
        case DataType::TOKENIZE(ctype):                                 \
            swap<ctype>(name, pvManager, containerManager);             \
            break;

        switch(desc->dataType) {
            TYPE_TABLE(SWITCH_ENTRY);
        default:
            die("%s: cannot swap data: %s has None type.",
                makeName().c_str(), name.c_str());
        }

#undef SWITCH_ENTRY        
    }
}

std::string PrimaryCellList::makeName() const
{
    return "Primary " + CellList::makeName();
}
