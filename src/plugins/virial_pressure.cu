#include "hip/hip_runtime.h"
#include "virial_pressure.h"
#include "utils/simple_serializer.h"

#include <core/datatypes.h>
#include <core/pvs/particle_vector.h>
#include <core/pvs/views/pv.h>
#include <core/simulation.h>
#include <core/utils/folders.h>
#include <core/utils/common.h>
#include <core/utils/cuda_common.h>
#include <core/utils/kernel_launch.h>

namespace VirialPressureKernels
{
__global__ void totalPressure(PVview view, const Stress *stress, FieldDeviceHandler region, VirialPressure::ReductionType *pressure)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    VirialPressure::ReductionType P = 0;
    Particle p;

    if (tid < view.size) {
        const Stress s = stress[tid];
        p.readCoordinate(view.particles, tid);

        if (region(p.r) > 0)
            P = (s.xx + s.yy + s.zz) / 3.0;
    }
    
    P = warpReduce(P, [](VirialPressure::ReductionType a, VirialPressure::ReductionType b) { return a+b; });

    if (__laneid() == 0)
        atomicAdd(pressure, P);
}
} // namespace VirialPressureKernels

VirialPressurePlugin::VirialPressurePlugin(const YmrState *state, std::string name, std::string pvName,
                                           FieldFunction func, float3 h, int dumpEvery) :
    SimulationPlugin(state, name),
    pvName(pvName),
    dumpEvery(dumpEvery),
    region(state, "field_"+name, func, h)
{}

VirialPressurePlugin::~VirialPressurePlugin() = default;

void VirialPressurePlugin::setup(Simulation* simulation, const MPI_Comm& comm, const MPI_Comm& interComm)
{
    SimulationPlugin::setup(simulation, comm, interComm);

    pv = simulation->getPVbyNameOrDie(pvName);

    region.setup(comm);

    info("Plugin %s initialized for the following particle vector: %s", name.c_str(), pvName.c_str());
}

void VirialPressurePlugin::handshake()
{
    SimpleSerializer::serialize(sendBuffer, pvName);
    send(sendBuffer);
}

void VirialPressurePlugin::afterIntegration(hipStream_t stream)
{
    if (state->currentStep % dumpEvery != 0 || state->currentStep == 0) return;

    PVview view(pv, pv->local());
    const Stress *stress = pv->local()->extraPerParticle.getData<Stress>(ChannelNames::stresses)->devPtr();

    localVirialPressure.clear(stream);
    
    SAFE_KERNEL_LAUNCH(
        VirialPressureKernels::totalPressure,
        getNblocks(view.size, 128), 128, 0, stream,
        view, stress, region.handler(), localVirialPressure.devPtr() );

    localVirialPressure.downloadFromDevice(stream, ContainersSynch::Synch);
    
    savedTime = state->currentTime;
    needToSend = true;
}

void VirialPressurePlugin::serializeAndSend(hipStream_t stream)
{
    if (!needToSend) return;

    debug2("Plugin %s is sending now data", name.c_str());

    waitPrevSend();
    SimpleSerializer::serialize(sendBuffer, savedTime, localVirialPressure[0]);
    send(sendBuffer);
    
    needToSend = false;
}

//=================================================================================

VirialPressureDumper::VirialPressureDumper(std::string name, std::string path) :
    PostprocessPlugin(name),
    path(path)
{
    if (std::is_same<VirialPressure::ReductionType, float>::value)
        mpiReductionType = MPI_FLOAT;
    else if (std::is_same<VirialPressure::ReductionType, double>::value)
        mpiReductionType = MPI_DOUBLE;
    else
        die("Incompatible type");
}

VirialPressureDumper::~VirialPressureDumper()
{
    if (activated)
        fclose(fdump);
}

void VirialPressureDumper::setup(const MPI_Comm& comm, const MPI_Comm& interComm)
{
    PostprocessPlugin::setup(comm, interComm);
    activated = createFoldersCollective(comm, path);
}

void VirialPressureDumper::handshake()
{
    auto req = waitData();
    MPI_Check( MPI_Wait(&req, MPI_STATUS_IGNORE) );
    recv();

    std::string pvName;
    SimpleSerializer::deserialize(data, pvName);

    if (activated)
    {
        auto fname = path + "/" + pvName + ".txt";
        fdump = fopen(fname.c_str(), "w");
        if (!fdump) die("Could not open file '%s'", fname.c_str());
        fprintf(fdump, "# time Pressure\n");
    }
}

void VirialPressureDumper::deserialize(MPI_Status& stat)
{
    TimeType curTime;
    VirialPressure::ReductionType localPressure, totalPressure;

    SimpleSerializer::deserialize(data, curTime, localPressure);

    if (!activated) return;

    MPI_Check( MPI_Reduce(&localPressure, &totalPressure, 1, mpiReductionType, MPI_SUM, 0, comm) );

    fprintf(fdump, "%g %.6e\n", curTime, totalPressure);
}

