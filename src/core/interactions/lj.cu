#include "lj.h"
#include "pairwise.impl.h"
#include "pairwise_interactions/lj.h"
#include "pairwise_interactions/lj_object_aware.h"

#include <core/celllist.h>

#include <memory>

InteractionLJ::InteractionLJ(const YmrState *state, std::string name, float rc, float epsilon, float sigma, float maxForce, bool objectAware, bool allocate) :
    Interaction(state, name, rc),
    objectAware(objectAware)
{
    if (!allocate) return;

    if (objectAware) {
        PairwiseLJObjectAware lj(rc, epsilon, sigma, maxForce);
        impl = std::make_unique<InteractionPair<PairwiseLJObjectAware>> (state, name, rc, lj);
    }
    else {
        PairwiseLJ lj(rc, epsilon, sigma, maxForce);
        impl = std::make_unique<InteractionPair<PairwiseLJ>> (state, name, rc, lj);
    }
}

InteractionLJ::InteractionLJ(const YmrState *state, std::string name, float rc, float epsilon, float sigma, float maxForce, bool objectAware) :
    InteractionLJ(state, name, rc, epsilon, sigma, maxForce, objectAware, true)
{}

InteractionLJ::~InteractionLJ() = default;

void InteractionLJ::setPrerequisites(ParticleVector *pv1, ParticleVector *pv2, CellList *cl1, CellList *cl2)
{
    impl->setPrerequisites(pv1, pv2, cl1, cl2);
}

std::vector<Interaction::InteractionChannel> InteractionLJ::getFinalOutputChannels() const
{
    return impl->getFinalOutputChannels();
}

void InteractionLJ::local(ParticleVector *pv1, ParticleVector *pv2,
                          CellList *cl1, CellList *cl2,
                          hipStream_t stream)
{
    impl->local(pv1, pv2, cl1, cl2, stream);
}

void InteractionLJ::halo(ParticleVector *pv1, ParticleVector *pv2,
                         CellList *cl1, CellList *cl2,
                         hipStream_t stream)
{
    impl->halo(pv1, pv2, cl1, cl2, stream);
}

void InteractionLJ::setSpecificPair(ParticleVector* pv1, ParticleVector* pv2, 
                                    float epsilon, float sigma, float maxForce)
{
    if (objectAware) {
        PairwiseLJObjectAware lj(rc, epsilon, sigma, maxForce);
        auto ptr = static_cast< InteractionPair<PairwiseLJObjectAware>* >(impl.get());
        ptr->setSpecificPair(pv1->name, pv2->name, lj);
    }
    else {
        PairwiseLJ lj(rc, epsilon, sigma, maxForce);
        auto ptr = static_cast< InteractionPair<PairwiseLJ>* >(impl.get());
        ptr->setSpecificPair(pv1->name, pv2->name, lj);
    }
}

